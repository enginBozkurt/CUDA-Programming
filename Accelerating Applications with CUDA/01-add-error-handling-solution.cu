
#include <hip/hip_runtime.h>
#include <stdio.h>

void init(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    a[i] = i;
  }
}

__global__
void doubleElements(int *a, int N)
{

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;

  /*
   * The previous code (now commented out) attempted
   * to access an element outside the range of `a`.
   */

  // for (int i = idx; i < N + stride; i += stride)
  for (int i = idx; i < N; i += stride)
  {
    a[i] *= 2;
  }
}

bool checkElementsAreDoubled(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    if (a[i] != i*2) return false;
  }
  return true;
}

int main()
{
  int N = 10000;
  int *a;

  size_t size = N * sizeof(int);
  hipMallocManaged(&a, size);

  init(a, N);

  /*
   * The previous code (now commented out) attempted to launch
   * the kernel with more than the maximum number of threads per
   * block, which is 1024.
   */

  size_t threads_per_block = 1024;
  /* size_t threads_per_block = 2048; */
  size_t number_of_blocks = 32;

  hipError_t syncErr, asyncErr;

  doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);

  /*
   * Catch errors for both the kernel launch above and any
   * errors that occur during the asynchronous `doubleElements`
   * kernel execution.
   */

  syncErr = hipGetLastError();
  asyncErr = hipDeviceSynchronize();

  /*
   * Print errors should they exist.
   */

  if (syncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(syncErr));
  if (asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));

  bool areDoubled = checkElementsAreDoubled(a, N);
  printf("All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");

  hipFree(a);
}
