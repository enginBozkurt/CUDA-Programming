
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void initializeElementsTo(int initialValue, int *a, int N)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < N)
  {
    a[i] = initialValue;
  }
}

int main()
{
  /*
   * Do not modify `N`.
   */

  int N = 1000;

  int *a;
  size_t size = N * sizeof(int);

  hipMallocManaged(&a, size);

  /*
   * Assume we have reason to want the number of threads
   * fixed at `256`: do not modify `threads_per_block`.
   */

  size_t threads_per_block = 256;

  /*
   * The following is idiomatic CUDA to make sure there are at
   * least as many threads in the grid as there are `N` elements.
   */

  size_t number_of_blocks = (N + threads_per_block - 1) / threads_per_block;

  int initialValue = 6;

  initializeElementsTo<<<number_of_blocks, threads_per_block>>>(initialValue, a, N);
  hipDeviceSynchronize();

  /*
   * Check to make sure all values in `a`, were initialized.
   */

  for (int i = 0; i < N; ++i)
  {
    if(a[i] != initialValue)
    {
      printf("FAILURE: target value: %d\t a[%d]: %d\n", initialValue, i, a[i]);
      exit(1);
    }
  }
  printf("SUCCESS!\n");

  hipFree(a);
}
