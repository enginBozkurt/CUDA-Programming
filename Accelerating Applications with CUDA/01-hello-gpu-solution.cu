
#include <hip/hip_runtime.h>
#include <stdio.h>

void helloCPU()
{
  printf("Hello from the CPU.\n");
}

/*
 * The addition of `__global__` signifies that this function
 * should be launced on the GPU.
 */

__global__ void helloGPU()
{
  printf("Hello from the GPU.\n");
}

int main()
{
  helloCPU();


  /*
   * Add an execution configuration with the <<<...>>> syntax
   * will launch this function as a kernel on the GPU.
   */

  helloGPU<<<1, 1>>>();

  /*
   * `cudaDeviceSynchronize` will block the CPU stream until
   * all GPU kernels have completed.
   */

  hipDeviceSynchronize();
}
