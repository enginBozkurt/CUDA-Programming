
#include <hip/hip_runtime.h>
#include <stdio.h>


/*
 * Notice the absence of the previously expected argument `N`.
 */

__global__ void loop()
{
  /*
   * This kernel does the work of only 1 iteration
   * of the original for loop. Indication of which
   * "iteration" is being executed by this kernel is
   * still available via `threadIdx.x`.
   */

  printf("This is iteration number %d\n", threadIdx.x);
}

int main()
{
  /*
   * It is the execution context that sets how many "iterations"
   * of the "loop" will be done.
   */

  loop<<<1, 10>>>();
  hipDeviceSynchronize();
}
