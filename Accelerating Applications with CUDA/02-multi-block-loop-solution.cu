
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void loop()
{
  /*
   * This idiomatic expression gives each thread
   * a unique index within the entire grid.
   */

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  printf("%d\n", i);
}

int main()
{
  /*
   * Additional execution configurations that would
   * work and meet the exercises contraints are:
   *
   * <<<5, 2>>>
   * <<<10, 1>>>
   */

  loop<<<2, 5>>>();
  hipDeviceSynchronize();
}
