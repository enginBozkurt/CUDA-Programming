#include "hip/hip_runtime.h"
#include <stdio.h>

#define N 2048 * 2048 // Number of elements in each vector

__global__ void saxpy(int * a, int * b, int * c)
{
  // Determine our unique global thread ID, so we know which element to process
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  
  for (int i = tid; i < N; i += stride)
    c[i] = 2 * a[i] + b[i];
}

int main()
{
  int *a, *b, *c;

  int size = N * sizeof (int); // The total number of bytes per vector

  int deviceId;
  int numberOfSMs;

  hipGetDevice(&deviceId);
  hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

  // Allocate memory
  hipMallocManaged(&a, size);
  hipMallocManaged(&b, size);
  hipMallocManaged(&c, size);

  // Initialize memory
  for( int i = 0; i < N; ++i )
  {
    a[i] = 2;
    b[i] = 1;
    c[i] = 0;
  }

  hipMemPrefetchAsync(a, size, deviceId);
  hipMemPrefetchAsync(b, size, deviceId);
  hipMemPrefetchAsync(c, size, deviceId);

  int threads_per_block = 256;
  int number_of_blocks = numberOfSMs * 32;

  saxpy <<<number_of_blocks, threads_per_block>>>( a, b, c );

  hipDeviceSynchronize(); // Wait for the GPU to finish

  // Print out the first and last 5 values of c for a quality check
  for( int i = 0; i < 5; ++i )
    printf("c[%d] = %d, ", i, c[i]);
  printf ("\n");
  for( int i = N-5; i < N; ++i )
    printf("c[%d] = %d, ", i, c[i]);
  printf ("\n");

  // Free all our allocated memory
  hipFree( a ); hipFree( b ); hipFree( c );
}
