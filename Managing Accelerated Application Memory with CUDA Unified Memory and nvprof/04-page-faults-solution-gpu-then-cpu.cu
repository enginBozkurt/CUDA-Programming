
#include <hip/hip_runtime.h>
__global__
void deviceKernel(int *a, int N)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = idx; i < N; i += stride)
  {
    a[i] = 1;
  }
}

void hostFunction(int *a, int N)
{
  for (int i = 0; i < N; ++i)
  {
    a[i] = 1;
  }
}

int main()
{
  int N = 2<<24;
  size_t size = N * sizeof(int);
  int *a;
  hipMallocManaged(&a, size);
  deviceKernel<<<256, 256>>>(a, N);
  hipDeviceSynchronize();
  hostFunction(a, N);
  hipFree(a);
}
